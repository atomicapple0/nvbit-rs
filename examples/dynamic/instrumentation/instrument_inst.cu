#include "hip/hip_runtime.h"
#include <cstdarg>
#include <stdint.h>
#include <stdio.h>

// does not work
/* #include "nvbit_tool.h" */

#include "utils/channel.hpp"
#include "utils/utils.h"

// contains definition of the mem_access_t structure
#include "common.h"

// Instrumentation function that we want to inject.
// Please note the use of extern "C" __device__ __noinline__
// to prevent "dead"-code elimination by the compiler.
extern "C" __device__ __noinline__ void instrument_inst(
    int pred, int opcode_id, uint64_t addr, uint64_t grid_launch_id,
    uint64_t pchannel_dev
    /* int pred, int opcode_id, int32_t vpc, bool is_mem, uint64_t addr, */
    /* int32_t width, int32_t desReg, int32_t srcReg1, int32_t srcReg2, */
    /* int32_t srcReg3, int32_t srcReg4, int32_t srcReg5, int32_t srcNum, */
    /* uint64_t pchannel_dev, uint64_t ptotal_dynamic_instr_counter, */
    /* uint64_t preported_dynamic_instr_counter, uint64_t pstop_report */
) {

  /* if thread is predicated off, return */
  if (!pred) {
    return;
  }

  int active_mask = __ballot_sync(__activemask(), 1);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  mem_access_t ma;

  /* collect memory address information from other threads */
  for (int i = 0; i < 32; i++) {
    ma.addrs[i] = __shfl_sync(active_mask, addr, i);
  }

  int4 cta = get_ctaid();
  ma.grid_launch_id = grid_launch_id;
  ma.cta_id_x = cta.x;
  ma.cta_id_y = cta.y;
  ma.cta_id_z = cta.z;
  ma.warp_id = get_warpid();
  ma.opcode_id = opcode_id;

  /* first active lane pushes information on the channel */
  if (first_laneid == laneid) {
    /* ChannelDev* channel_dev = (ChannelDev*)pchannel_dev; */
    /* channel_devn->push(&ma, sizeof(mem_access_t)); */
    ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
    channel_dev->push(&ma, sizeof(mem_access_t));
  }

  /* const int active_mask = __ballot_sync(__activemask(), 1); */
  /* const int predicate_mask = __ballot_sync(__activemask(), pred); */
  /* const int laneid = get_laneid(); */
  /* const int first_laneid = __ffs(active_mask) - 1; */

  /* if ((*((bool *)pstop_report))) { */
  /*   if (first_laneid == laneid) { */
  /*     atomicAdd((unsigned long long *)ptotal_dynamic_instr_counter, 1); */
  /*     return; */
  /*   } */
  /* } */

  /* inst_trace_t ma; */

  /* if (is_mem) { */
  /*   // collect memory address information */
  /*   for (int i = 0; i < 32; i++) { */
  /*     ma.addrs[i] = __shfl_sync(active_mask, addr, i); */
  /*   } */
  /*   ma.width = width; */
  /*   ma.is_mem = true; */
  /* } else { */
  /*   ma.is_mem = false; */
  /* } */

  /* int4 cta = get_ctaid(); */
  /* int unique_thread_id = threadIdx.z * blockDim.y * blockDim.x + */
  /*                        threadIdx.y * blockDim.x + threadIdx.x; */
  /* ma.warpid_tb = unique_thread_id / 32; */

  /* ma.cta_id_x = cta.x; */
  /* ma.cta_id_y = cta.y; */
  /* ma.cta_id_z = cta.z; */
  /* ma.warpid_sm = get_warpid(); */
  /* ma.opcode_id = opcode_id; */
  /* ma.vpc = vpc; */
  /* ma.GPRDst = desReg; */
  /* ma.GPRSrcs[0] = srcReg1; */
  /* ma.GPRSrcs[1] = srcReg2; */
  /* ma.GPRSrcs[2] = srcReg3; */
  /* ma.GPRSrcs[3] = srcReg4; */
  /* ma.GPRSrcs[4] = srcReg5; */
  /* ma.numSrcs = srcNum; */
  /* ma.active_mask = active_mask; */
  /* ma.predicate_mask = predicate_mask; */
  /* ma.sm_id = get_smid(); */

  /* // first active lane pushes information on the channel */
  /* if (first_laneid == laneid) { */
  /*   ChannelDev *channel_dev = (ChannelDev *)pchannel_dev; */
  /*   channel_dev->push(&ma, sizeof(inst_trace_t)); */
  /* } */
}
